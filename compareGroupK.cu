#include "hip/hip_runtime.h"

#include "compareGroupK.h"

int compareGroupK(int* blockScore, LinkedList *h_aligned, int* done, int* r, int* moreCents, int* cents, int* numCents, intLinkedList *tracer){
	//flag to indicate first run through while
	int flag=1;
	//runs until all distinct centroids are found
	while(1){
		int i, iflag=0;
		int max_similarity=0;
		for(i=0; i<*r; i++){ // find max i & max j
			//increments i to a spot previous centroids have a score of 0 at this point
			while(*(moreCents+i)!=-1){
				i++;
				//at end break out of while and for i
				if(i==*r){
					iflag=1;
					break;
				}
			}
			if(iflag==1){
				break;
			}
			int similarity=0, itmp, jtmp=-1, j, jflag=0;
			for(j=i+1; j<*r; j++){
				//increments j to a spot previous centroids have a score of 0 at this point
				while(*(moreCents+j)!=-1) {
					j++;
					//at end break out of while and for j
					if(j==*r){
						jflag=1;
						break;
					}
				}
				if(jflag==1)
					break;
				if(*(blockScore+*r*i+j) > 0&&flag==1){
					(*(done+i))++;
					(*(done+j))++;
				}
				//updates similarity to highest blockScore[i][?]
				if(*(blockScore+*r*i+j) > similarity){
					itmp=i;
					jtmp=j;
					similarity=*ptrMath2D(blockScore, i, j, *r);
				}
			}
			//no similar groups to i and on first iteration
			//done[i] and done[jtmp] are incremented
			/*if(similarity!=0&&flag==1){
				(*(done+i))++;
				if(jtmp != -1){
					(*(done+jtmp))++;
				}
			}*/
			//updates max_similarity, puts values in cents
			if(similarity>max_similarity){
				//max_i=itmp;
				//max_j=jtmp;
				*(cents+*numCents*2+0)=itmp;
				*(cents+*numCents*2+1)=jtmp;
				max_similarity=similarity;
			}
		}
		//increments moreCents where the centroid has a score of nonzero and the centroid itself
		if(max_similarity != 0){
			flag=0;
			printf("Merge groups at indicies (%d, %d) with a score of: %d\n", *(cents+*numCents*2+0), *(cents+*numCents*2+1), max_similarity);
			for(i=0; i<*r; i++){
				if(*(blockScore+*r*(*(cents+*numCents*2+0))+i)>0||*(blockScore+*r*i+(*(cents+*numCents*2+0)))>0){
					(*(moreCents+i))++;
				}
				else if(i==*(cents+*numCents*2+0)){
					(*(moreCents+i))++;
				}
			}
		}
		//no Centroids, max_similarity is 0, done with main while
		else if(*numCents==0)
			return -1;//break;
		//max_similarity is 0, done finding centroids
		else
			return 1;
		//merges centroid and match
		Node *cur = h_aligned->root;
		NodeI *curT = tracer->root;
		Node *previous;
		NodeI *previousT;
		int cur_pos = 1;
		// ITERATE TO MATCH
		while(cur_pos < *(cents+*numCents*2+1)+1){
			//accounts for centroids already removed in list
			for(i=0; i<*numCents; i++){
				//if(cur_pos==*(cents+i*2+0)||cur_pos==*(cents+i*2+1)){
				if(cur_pos==*(cents+i*2+1)){
					cur_pos++;
				}
			}
			previous = cur;
			previousT = curT;
			cur = cur->next;
			curT = curT->next;
			cur_pos += 1;
		}
		//removes match
		CharNode *group = cur->data;
		intNodeI *groupT = curT->data;
		//previous->next = cur->next;
		//previousT->next = curT->next;
			if(cur == h_aligned->root){
				h_aligned->root = cur->next;
				tracer->root = curT->next;
			}
			else{
                		previous->next = cur->next;
				previousT->next = curT->next;
			}
		free(cur);
		free(curT);
		h_aligned->num_elems--;
		tracer->num_elems--;
		//(*r) = h_aligned->num_elems;

		// ITERATE TO CENTROID & GROUP
		cur_pos = 1;
		Node *current = h_aligned->root;
		NodeI *currentT = tracer->root;
		while(cur_pos < *(cents+*numCents*2+0)){
			//accounts for centroids already removed in list
			for(i=0; i<*numCents; i++){
				//if(cur_pos==*(cents+i*2+0)||cur_pos==*(cents+i*2+1)){
				if(cur_pos==*(cents+i*2+1)) {
					cur_pos++;
				}
			}
			current = current->next;
			currentT = currentT->next;
			cur_pos += 1;
		}
		//groups centroid and match
		CharNode *c = current->data;
		intNodeI *cT = currentT->data;;
		while(c->next != NULL){
			c = c->next;
			cT = cT->next;
		}
		c->next = group;
		cT->next = groupT;
		//increments number of centroids
		(*numCents)++;
	}
}

void fuzzyK(int* blockScore, LinkedList *h_aligned, int* r, int* numCents, int* fuzzymatch, int* cents, int* fuzziness, int* fuzzies, intLinkedList *tracer, int* tolerance){
	int m, f;
			/*printf("**********************************************************************************\n");
			printf("* OUR LIST BEFORE ANYTHING                                                        \n");
			List_print(h_aligned, 1126);
			printf("**********************************************************************************\n");*/
	//printf("running fuzzy\n");
	//each iteration populates fuzzymatch[centroid][m]
	for(m=0; m<*fuzziness; m++){
		//printf("first for\n");
		int k;
		//finds a fuzzy match for each centroid
		for(k=0; k< *numCents; k++){
			//printf("second for k:%d numCents:%d\n", k, *numCents);
			int i;
			int max_simlow=0, match=-1, cent, breakflag;
			//finds the highest degree of membership
			for(i=0; i< *numCents; i++){
				breakflag=0;
				//printf("numcent for i:%d numCents:%d\n", i, *numCents);
				//checks that no fuzzymatch has been found for this spot
				while(*ptrMath2D(fuzzymatch, i, m, *fuzziness)!=-1){
					i++;
					//printf("i increased\n");
					if(i==*numCents){
						breakflag=1;
						break;
					}
				}
				if(breakflag==1)
					break;
				int j;
				int simlow=0, bposi, bposj, bposfuzzy, matchtmp, simlowtmp=0, fuzzyflag=0;
				//computes finds the highest low blockScore and the match for this centroid
				for(j=0; j<*r; j++){
					fuzzyflag=0;
					//printf("j:%d r:%d cent1:%d\n", j, *r, *ptrMath2D(cents, i, 0, 2));
					//assigns correct spot
					int b=0, bflag=0;
					while(b==0){
						b=1;
						int n;
						for(n=0; n<*numCents; n++){
                                        		//accounts for centroid matches already removed
                                        		if(j==*ptrMath2D(cents, n, 1, 2)||j==*ptrMath2D(cents, n, 0, 2)){
                                                		j++;
								b=0;
								//printf("j increased\n");
							}
                                        		//accounts for fuzzy matches already removed
                                        		for(f=0; f<m+1; f++){
								if(*ptrMath2D(fuzzymatch, n, f, *fuzziness)==-1)
									break;
                                                		else if(j==*ptrMath2D(fuzzymatch, n, f, *fuzziness)){
                                                        		j++;
									b=0;
									//printf("j increased\n");
								}
                                        		}
                                        		if(j >= *r){
								b=1;
								bflag=1;
								break;
							}
                                		}
					}
					if(bflag==1){
						//printf("breaking at bflag\n");
						break;
					}
					if(*ptrMath2D(cents, i, 0, 2)>j){
						bposi=*r*j+*ptrMath2D(cents, i, 0, 2);
						//printf("bposi:[%d][%d]\n", j, *ptrMath2D(cents, i, 0, 2));
					}
					else if(*ptrMath2D(cents, i, 0, 2)<j){
						bposi=*r*(*ptrMath2D(cents, i, 0, 2))+j;
						//printf("bposi:[%d][%d]\n", *ptrMath2D(cents, i, 0, 2), j);
					}
					//check that it's not 0, otherwise sets the flag (all matches must be non-zero)
					if(*(blockScore+bposi)!=0){
						simlowtmp=*(blockScore+bposi);
						//printf("simlowtmp1:%d\n", simlowtmp);
					}
					else
						fuzzyflag=1;
					if(*ptrMath2D(cents, i, 1, 2)>j){
                                	        bposj=*r*j+*ptrMath2D(cents, i, 1, 2);
						//printf("bposj:[%d][%d]\n", j, *ptrMath2D(cents, i, 1, 2));
					}
                                	else if(*ptrMath2D(cents, i, 1, 2)<j){
                                	        bposj=*r*(*ptrMath2D(cents, i, 1, 2))+j;
						//printf("bposj:[%d][%d]\n", *ptrMath2D(cents, i, 1, 2), j);
					}
					if(*(blockScore+bposj)!=0){
						//printf("nonzero\n");
						if(*(blockScore+bposj)<simlowtmp){
							simlowtmp=*(blockScore+bposj);
							//printf("simlowtmp2:%d\n", simlowtmp);
						}
					}
					else
						fuzzyflag=1;

					int f;
					//does the same as above for previously found fuzzy nodes
					for(f=1; f<m+1; f++){
						//printf("fuzzy for %d\n", f);
						if(*ptrMath2D(fuzzymatch, i, f-1, *fuzziness)!=-1){
							if(*ptrMath2D(fuzzymatch, i, f-1, *fuzziness)>j){
								bposfuzzy=*r*j+*ptrMath2D(fuzzymatch, i, f-1, *fuzziness);
								//printf("bposfuzzy:(%d, %d) i:%d\n", j, *ptrMath2D(fuzzymatch, i, f-1, *fuzziness), i);
							}
							else if(*ptrMath2D(fuzzymatch, i, f-1, *fuzziness)<j){
								bposfuzzy=(*r)*(*ptrMath2D(fuzzymatch, i, f-1, *fuzziness))+j;
								//printf("bposfuzzy:(%d, %d) i:%d\n", *ptrMath2D(fuzzymatch, i, f-1, *fuzziness), j, i);
							}
							//printf("in middle\n");
							if(*(blockScore+bposfuzzy)!=0 ) {
								//printf("Score:%d simlow: simlowtmp: %d\n", *(blockScore+bposfuzzy), simlowtmp);
								//printf("error\n");
								if(*(blockScore+bposfuzzy)<simlowtmp)
									simlowtmp=*(blockScore+bposfuzzy);
							}
							else
								fuzzyflag=1;
						}
						else
							break;
					}
					//printf("after fuzzy for\n");
					//updates sim low if all are non-zero scores
					if(fuzzyflag==0&&simlowtmp>simlow){
						//printf("simlow:%d\n", simlowtmp);
						simlow=simlowtmp;
						matchtmp=j;
						//printf("matchtmp:%d\n", matchtmp);
					}
				}



				float percentage = ((float)simlow/(float)(*ptrMath2D(blockScore, *ptrMath2D(cents, i, 0, 2), *ptrMath2D(cents, i, 1, 2), *r)));
				//updates max_simlow and match(highest match found)
				if(simlow>max_simlow && *tolerance<=100*percentage){
					cent=i;
					match=matchtmp;
					max_simlow=simlow;
					//printf("found! max_simlow:%d match:%d\n", max_simlow, match);
				}
			}
			//match found! updates fuzzymatch[cent][m] and increments fuzzies
			if(match!=-1){
				printf("Merging %d with centroid#: %d at position %d with low score of %d\n", match, cent, *ptrMath2D(cents, cent, 0, 2), max_simlow);
				*ptrMath2D(fuzzymatch, cent, m, *fuzziness)=match;
				(*fuzzies)++;
			}
			//no match found at highest degree of fuzziness desired, done
			else if(m==*fuzziness-1){
				//printf("return\n");
				return;
			}
			//no match found at this degree of fuzziness, may be some at higher level
			else{
				//printf("break\n");
				break;
			}
			//merges centroid and match
                	Node *cur = h_aligned->root;
			NodeI *curT = tracer->root;
                	Node *previous;
			NodeI *previousT;
                	int cur_pos = 0;
                	// ITERATE TO MATCH
			int monkeyflag=0;
			//printf("going to: %d\n", *ptrMath2D(fuzzymatch, cent, m, *fuzziness));
                	while(cur_pos < *ptrMath2D(fuzzymatch, cent, m, *fuzziness)){
				//accounts for centroids already removed in list
                	        //printf("current cur_pos:%d\n", cur_pos);
				int b=0;
				while(b==0){
					b=1;
					for(i=0; i<*numCents; i++){
						//accounts for centroid matches already removed
                	                	//printf("i:%d numCents:%d centmatch:%d\n", i, *numCents, *ptrMath2D(cents, i, 1, 2));
						if(cur_pos==*ptrMath2D(cents, i, 1, 2)){
							//printf("CURPOS IS %d, SO INCREMENTING: %d\n", cur_pos, *ptrMath2D(cents, i, 1, 2));
							//printf("pos++\n");
							b=0;
                	                        	cur_pos++;
							if(cur_pos >= *ptrMath2D(fuzzymatch, cent, m, *fuzziness)){
								b=1;
								monkeyflag=1;
								//printf("break1\n");
								break;
							}
						}
						//accounts for fuzzy matches already removed
						for(f=0; f<m+1; f++){
							if(cur_pos==*ptrMath2D(fuzzymatch, i, f, *fuzziness)&&(i!=cent || f!=m)){
							//printf("CURPOS IS %d, SO INCREMENTING: %d\n", cur_pos, *ptrMath2D(fuzzymatch, i, f, *fuzziness));
								//printf("pos++");
								b=0;
								cur_pos++;
								if(cur_pos >= *ptrMath2D(fuzzymatch, cent, m, *fuzziness)){
									b=1;
									monkeyflag=1;
									//printf("break2\n");
									break;
								}
							}
							else if(*ptrMath2D(fuzzymatch, i, f, *fuzziness)==-1&&(i!=cent || f!=m))
								break;
							else {
							//printf("CURPOS IS %d, NOT INCREMENTING: %d.  I: %d  CENT: %d, F: %d, M: %d\n", cur_pos, *ptrMath2D(fuzzymatch, i, f, *fuzziness), i, cent, f, m);

							}
						}
						//if(cur_pos >= *ptrMath2D(fuzzymatch, cent, m, *fuzziness)-1){
						//	printf("ever running?\n");
						//	b=1;
						//	monkeyflag=1;
						//	break;
						//}
                	        	}
				}
				if(monkeyflag==1)
					break;
                	        previous = cur;
				previousT = curT;
				//printf("JUMPING CUR\n");
                	        cur = cur->next;
				curT = curT->next;
                	        cur_pos += 1;
                	}
			/*printf("end of loop\n");
			printf("**********************************************************************************\n");
			printf("* OUR LIST AFTER FIRST MERGES                                                     \n");
			List_print(h_aligned, 1126);
			printf("**********************************************************************************\n");*/
			//printf("cur_pos: %d\n", cur_pos);
                	//removes match
			//printf("%d\n", (cur == NULL));
			//printf("%d\n", (cur->data == NULL));
                	CharNode *group = cur->data;
			//printf("group assigned\n");
			intNodeI *groupT = curT->data;
                	//previous->next = cur->next;
			//previousT->next = curT->next;
			if(cur == h_aligned->root){
				h_aligned->root = cur->next;
				tracer->root = curT->next;
			}
			else{
                		previous->next = cur->next;
				previousT->next = curT->next;
			}
                	free(cur);
			free(curT);
                	h_aligned->num_elems--;
			tracer->num_elems--;

                	// ITERATE TO CENTROID & GROUP
                	cur_pos = 0;
                	Node *current = h_aligned->root;
			NodeI *currentT = tracer->root;

			//printf("end of assignments\n");
			//printf("CENTROID GOING TO: %d", *ptrMath2D(cents, cent, 0, 2));
			monkeyflag=0;
			while(cur_pos < *ptrMath2D(cents, cent, 0, 2)){
				//printf("centroid while\n");
				int b=0;
				while(b==0){
					b=1;
					for(i=0; i<*numCents; i++){
			//printf("CENTROID %d SECOND VALUE: %d\n", i, *ptrMath2D(cents, i,  1, 2));
                                        	if(cur_pos==*ptrMath2D(cents, i, 1, 2)){
							//printf("CURPOS IS %d, SO INCREMENTING", cur_pos, *ptrMath2D(cents, i, 1, 2));
                                                	b=0;
							cur_pos++;
						}
                                        	for(f=0; f<m+1; f++){
							//printf("FUZZY VALUE (%d, %d): %d\n", i, f, *ptrMath2D(fuzzymatch, i, f, *fuzziness));
                                                	if(cur_pos==*ptrMath2D(fuzzymatch, i, f, *fuzziness)){
							//printf("CURPOS IS %d, FUZZY SO INCREMENTING", cur_pos, *ptrMath2D(fuzzymatch, i, f, *fuzziness));
								b=0;
                                                        	cur_pos++;
							}
                                        	}
                                        	if(cur_pos >= *ptrMath2D(cents, cent, 0, 2)){
							//printf("CURPOS IS %d, WHICH IS >= %d, SO BREAKING", cur_pos, *ptrMath2D(cents, cent, 0, 2));
                                                	b=1;
							monkeyflag=1;
							break;
						}
                                	}
				}
				if(monkeyflag!=0)
					break;
				//printf("JUMPING CURRENT\n");
                	        current = current->next;
				//print_list(current->data, 1126);
				currentT = currentT->next;
                	        cur_pos++;
				//printf("CUR POS END OF LOOP: %d\n", cur_pos);
                	}
			//printf("cur_pos: %d\n", cur_pos);
                	//groups centroid and match
                	CharNode *c = current->data;
			intNodeI *cT = currentT->data;
                	while(c->next != NULL){
                	        c = c->next;
				cT = cT->next;
			}
                	c->next = group;
			cT->next = groupT;
			/*printf("**********************************************************************************\n");
			printf("* OUR LIST CENTROID MERGE                                                         \n");
			List_print(h_aligned, 1126);
			printf("**********************************************************************************\n");*/
		}
	}
}
