#include "hip/hip_runtime.h"
#include "compareGroup.h"

int compareGroup(int* blockScore, LinkedList *h_aligned, int *done, int* r, intLinkedList *tracer){
	int max_similarity=0, max_i, max_j, i;
	for(i=0; i<*r; i++){ // find max i & max j
		int similarity=0, itmp, jtmp=-1, j;
		for(j=i+1; j<*r; j++){
			if(*(blockScore+*r*i+j)>0){
				done[i]++;
				done[j]++;
			}
			if(*(blockScore+*r*i+j) > similarity){
				itmp=i;
				jtmp=j;
				similarity=*(blockScore+*r*i+j);
			}
		}
		/*if(similarity!=0){
			done[i]++;
			if(jtmp != -1){
				done[jtmp]++;
			}
		}*/
		if(similarity>max_similarity){
			max_i=itmp;
			max_j=jtmp;
			max_similarity=similarity;
			//printf("similarity: %i\n", similarity);
		}
//printf("%i\n", i);
	}
//printf("%d", max_similarity);
	if(max_similarity != 0)
		printf("Merge groups at indicies (%d, %d) with a score of: %d\n", max_i, max_j, max_similarity);
	else
		return -1;//break;

	Node *cur = h_aligned->root;
	NodeI *curT = tracer->root;
	Node *previous;
	NodeI *previousT;
	int cur_pos = 1;

	// ITERATE TO MAX J
	while(cur_pos < max_j+1){
		previous = cur;
		previousT = curT;
		cur = cur->next;
		curT = curT->next;
		cur_pos += 1;
	}

//printf("before charNode\n");
	CharNode *group = cur->data;
	intNodeI *groupT = curT->data;
	previous->next = cur->next;
	previousT->next = curT->next;
	free(cur);
	free(curT);
	h_aligned->num_elems--;
	tracer->num_elems--;
	*r = h_aligned->num_elems;

//printf("before iterate to max\n");
	// ITERATE TO MAX I & GROUP
	cur_pos = 1;
	Node *current = h_aligned->root;
	NodeI *currentT = tracer->root;
//printf("before while\n");
	while(cur_pos < max_i+1){
		current = current->next;
		currentT = currentT->next;
		cur_pos += 1;
	}
//printf("after while\n");
	CharNode *c = current->data;
	intNodeI *cT = currentT->data;
	while(c->next != NULL){
		c = c->next;
		cT = cT->next;
	}
	c->next = group;
	cT->next = groupT;
	return max_j;
}

