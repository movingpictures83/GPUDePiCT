#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <assert.h>
#include <stdio.h>
#include <string.h>
#include "LinkedList.h"
#include "intLinkedList.h"
#include <sys/time.h>
#include "ClusterCPU.h"
#include "Codons.h"
#include "LoadFile.h"
#include "NucCodons.h"
#include "flatten.h"
#include "compareGroup.h"
#include "compareGroupK.h"
#include "getprimer.h"
#include "getNucPrimer.h"
#include "pointerMath.h"
#include "Cluster.h"
/*  Degenerate Primer Design via Clustering,		  *
 *  on a GPU utilizing Nvidia's Cuda Platform		  *
 *  		  					  *
 *  Written by Philippe Novikov & James Parda		  *
 *  Summer Research 2012 with Dr. Trevor Cickovski        *
 *                                                        */
#include "GPUDePiCtPlugin.h"
#include <fstream>

void GPUDePiCtPlugin::input(std::string file) {
 inputfile = file;
 std::ifstream ifile(inputfile.c_str(), std::ios::in);
 while (!ifile.eof()) {
   std::string key, value;
   ifile >> key;
   ifile >> value;

   parameters[key] = value;
 }
 std::string composite = std::string(PluginManager::prefix())+"/"+parameters["inputfile"];
		filename = (char*) composite.c_str();
		ROWS = atoi(parameters["rows"].c_str());
		SEQ_LENGTH = atoi(parameters["seqlength"].c_str());
		boolVal = atoi(parameters["AAorNuc"].c_str());
		algorithm = atoi(parameters["kmeans"].c_str());
		fuzziness = atoi(parameters["fuzzy"].c_str());
		tolerance = atoi(parameters["tolerance"].c_str());
		if (parameters.count("minlength") != 0)
			MIN_PRIMER_LENGTH = atoi(parameters["minlength"].c_str());

	data = fopen(filename, "r");
	aligned = (char*) malloc(ROWS*SEQ_LENGTH*sizeof(char));

	// filetype checker and reader
	LoadFile(filename, aligned, data, ROWS, SEQ_LENGTH);
}

void GPUDePiCtPlugin::run() {
	//load correct set of codons
  	if(boolVal==0)
		insertCodons(simacids, codons);
	else
		insertNucCodons(simNucs, nucCodons);

	r = ROWS;
	int *done = (int*)malloc(ROWS*sizeof(int));


/* Cluster sequences */
  	//Start timer
  	struct timeval start, finish;
  	struct timezone tz;
  	gettimeofday(&start, &tz);

      	// Populate the list with our sequences
      	h_aligned = (LinkedList*)malloc(sizeof(LinkedList));
      	Node *root = (Node*)malloc(sizeof(Node));
      	CharNode *first_sequence = (CharNode*)malloc(sizeof(CharNode));
      	first_sequence->data = &aligned[0*SEQ_LENGTH+0];
	first_sequence->next = NULL;
      	root->data = first_sequence;
        root->next = NULL;
	intLinkedList *tracer = (intLinkedList*)malloc(sizeof(intLinkedList));
	tracer->root = NULL;
	for(x=0; x<ROWS; x++){
		intNodeI *xintnode=(intNodeI*)malloc(sizeof(intNodeI));
		xintnode->data=x;
		xintnode->next=NULL;
		NodeI *xnodeI =(NodeI*)malloc(sizeof(NodeI));
		xnodeI->data=xintnode;
		xnodeI->next=NULL;
		insertendI(tracer, xnodeI);
	}
	//List_printI(tracer);
	intLinkedList *final_tracer =(intLinkedList*)malloc(sizeof(intLinkedList));
        final_tracer->root = NULL;
	h_aligned->root = root;
        h_aligned->num_elems = 1;
        for(x=1; x<ROWS; x++) {
		int q;
		/*printf("INSERTING STRING: ");
		for (q = 0; q < SEQ_LENGTH; q++)
		   printf("%c", aligned[x*SEQ_LENGTH+q]);
		printf("\n");*/
                insert(h_aligned, x+1, &aligned[x*SEQ_LENGTH+0]);
        }
	groups = (int*) malloc(ROWS*sizeof(int));

	//initialize second list for when groups with a max similarity of 0 are removed
	final_list = (LinkedList*)malloc(sizeof(LinkedList));
	final_list->root = NULL;
	final_groups = (int*) malloc(ROWS*sizeof(int));
	for (i = 0; i < ROWS; i++)
		final_groups[i] = 0;
	final_list->num_elems=0;

	#ifndef CPU
		//declaration of gpu data structures
    		hipMalloc( (void**) &gpu_sim, (unsigned int)ROWS*ROWS*SEQ_LENGTH*sizeof(bool));
    		hipMalloc( (void**) &gpu_group, ROWS*sizeof(int));
    		//size_t size = SEQ_LENGTH*sizeof(char); // size of an alignment
		hipMalloc( (void**) &gpu_aligned, ROWS*SEQ_LENGTH*sizeof(char));

		//Define numBlocks and block dimensions for gpu operations
      		unsigned int numBlocks = ((unsigned int)ROWS*ROWS*SEQ_LENGTH - 1)/BLOCKSIZE+1;
      		unsigned int numBlocks2 =((unsigned int)ROWS*ROWS-1)/BLOCKSIZE+1;
		unsigned int numGrids=1;
		unsigned int numGridsY=1;
		unsigned int numGrids2=1;
		unsigned int numGrids2Y=1;
      		if (numBlocks > 65535){
         		numGrids = numBlocks / 65535;
         		if (numGrids % numBlocks != 0)
				numGrids++;
         		numBlocks = 65535;
			if (numGrids > 65535) {
				numGridsY = numGrids / 65535;
				if (numGridsY % numGrids != 0)
					numGridsY++;
				numGrids = 65535;
			}
      		}
		if (numBlocks2 > 65535) {
			numGrids2 = numBlocks2 / 65535;
			if (numGrids2 % numBlocks2 != 0)
				numGrids2++;
			numBlocks2 = 65535;
			if (numGrids2 > 65535) {
				numGrids2Y = numGrids2 / 65535;
				if (numGrids2Y % numGrids2 != 0)
					numGrids2Y++;
				numGrids2 = 65535;
			}
		}
		int BLOCKXY = BLOCKX*BLOCKY;
		dim3 dimBlock(BLOCKX, BLOCKY, BLOCKSIZE/BLOCKXY);
		//dim3 dimBlock(BLOCKSIZE/BLOCKXY, BLOCKX, BLOCKY);
		dim3 dimBlock2(BLOCKX, BLOCKY, BLOCKSIZE/BLOCKXY);
		//dim3 dimBlock2(BLOCKSIZE/BLOCKXY, BLOCKX, BLOCKY);
		dim3 dimGrid(65535, numGrids, numGridsY);
		dim3 dimGrid2(65535, numGrids2, numGrids2Y);
      		printf("%d grids of %d blocks have been allocated for this process.\n", numGrids, numBlocks);
		printf("%d blocks have been allocated for this process.\n",numBlocks2);
	#else
	   int* sim_mat = malloc((int)ROWS*ROWS*SEQ_LENGTH*sizeof(int));
	#endif
			/*printf("**********************************************************************************\n");
			printf("* OUR LIST IN MAIN                                                                \n");
			List_print(h_aligned, 1126);
			printf("**********************************************************************************\n");*/
	char* cpu_aligned = (char*)malloc(ROWS*SEQ_LENGTH*sizeof(char));
	//char cpu_aligned[ROWS][SEQ_LENGTH];
    	while(1){
		//flatten h_aligned and copy it to the gpu
		/*int a, c;;
		for (a = 0; a < ROWS; a++) {
			for (c = 0; c < SEQ_LENGTH; c++) {
				printf("%c ", aligned[a*SEQ_LENGTH+c]);
			}
			printf("\n");
		}*/
      		flatten(h_aligned->root, cpu_aligned, groups, ROWS, SEQ_LENGTH);
		#ifndef CPU
 	//			bool* cpu_sim = (bool*)malloc(ROWS*ROWS*SEQ_LENGTH*sizeof(bool));

			hipMemcpy(gpu_aligned, cpu_aligned, ROWS*SEQ_LENGTH*sizeof(char), hipMemcpyHostToDevice);
      			hipMemcpy(gpu_group, groups, ROWS*sizeof(int), hipMemcpyHostToDevice);
			//compute sim for either aa or nuc
			if(boolVal ==0)
				computeSim<<< dimGrid, dimBlock >>>( (bool (*))gpu_sim, (char (*))gpu_aligned, (int (*))gpu_group, ROWS, SEQ_LENGTH, r );
			else
				computeNucSim<<< dimGrid, dimBlock >>>( (bool (*))gpu_sim, (char (*))gpu_aligned, (int (*))gpu_group, ROWS, SEQ_LENGTH, r );
		//initialize blockScore(cpu) and bScore(gpu)

/*		hipMemcpy(cpu_sim, gpu_sim, ROWS*ROWS*SEQ_LENGTH*sizeof(bool), hipMemcpyDeviceToHost);
		int j, k;
				for (k=0; k<SEQ_LENGTH; k++){
					printf("%d", cpu_sim[0*ROWS*SEQ_LENGTH+1*SEQ_LENGTH+k]);
				}
				printf("\n");
				List_print(h_aligned, SEQ_LENGTH);
*/		
		#else
			//printf("COMPUTING SIM....\n");
			computeSimCPU(cpu_aligned, groups, r, ROWS, SEQ_LENGTH, sim_mat);
			/*int a, b, c;
			for (a = 0; a < ROWS; a++)
			for (b = 0; b < ROWS; b++)
			for (c = 0; c < SEQ_LENGTH; c++)
			   if (sim_mat[a*ROWS*SEQ_LENGTH+b*SEQ_LENGTH+c] != 0)
			      printf("NON-ZERO ENTRY FOUND\n");*/
			//printf("DONE....\n");
		#endif

		int* blockScore=(int*)malloc(r*r*sizeof(int));
		#ifndef CPU
			hipMalloc((void**)&bScore, r*r*sizeof(int));
			//printf("BEFORE BLOCK\n");
			//List_print(h_aligned, SEQ_LENGTH);
			//compute block score for nuc or aa, copy back to cpu and free gpu version
			//printf("ALLOCATED %d\n", r*r);
			if(boolVal ==0)
				computeBlock<<<dimGrid2, dimBlock2>>>((bool (*))gpu_sim, (int  (*))bScore, ROWS, SEQ_LENGTH, r, MIN_PRIMER_LENGTH);
      			else{
				//printf("COMPUTE NUC BLOCK %d\n", numBlocks2);
				computeNucBlock<<<dimGrid2, dimBlock2>>>((bool (*))gpu_sim, (int  (*))bScore, r, ROWS, SEQ_LENGTH, MIN_PRIMER_LENGTH);
      			}
			/*gettimeofday(&finish, &tz);
        		double block1Elapsed = ( finish.tv_sec - start.tv_sec ) * 1000.0 + ( finish.tv_usec - start.tv_usec ) / 1000.0;
        		printf("Time to before copy: %lf ms\n", block1Elapsed);*/

			hipMemcpy(blockScore, bScore, r*r*sizeof(int), hipMemcpyDeviceToHost);
			hipFree(bScore);
		#else
			//printf("R:%d\n", r);
			if(boolVal ==0)
				computeBlockCPU(sim_mat,/*cpu_aligned, groups,*/ blockScore, r, ROWS, SEQ_LENGTH, MIN_PRIMER_LENGTH);
			else
				computeNucBlockCPU(cpu_aligned, groups, blockScore, r, SEQ_LENGTH, MIN_PRIMER_LENGTH);
		#endif

		/*gettimeofday(&finish, &tz);
        	double block2Elapsed = ( finish.tv_sec - start.tv_sec ) * 1000.0 + ( finish.tv_usec - start.tv_usec ) / 1000.0;
        	printf("Time to after copy: %lf ms\n", block2Elapsed);*/
		/*int v, b;
		printf("    ");
		for(v=0; v<r; v++)
			printf("G%2d|", v);
		printf("\n");
		for(v=0; v<r; v++){
			printf("G%2d|", v);
			for(b=0; b<r; b++)
				printf("%3d|", *(blockScore+v*r+b));
			printf("\n");
		}*/


		//initialize arrays for centroids
		//int done[r];
		int moreCents[r];
		//int i;
		for(i=0; i<r; i++){
			moreCents[i]=-1;
			//done[i]=0;
		}
		numCents=0;
		int cents[r];
		for(i=0; i<r; i++){
			cents[i*2+0]=-1;
			cents[i*2+1]=-1;
		}

		//int *done = (int*)malloc(ROWS*sizeof(int));
		for (i = 0; i < ROWS; i++)
			done[i] = 0;

		//initializes arrays for fuzzy K
		int fuzzymatch[r*fuzziness];
		for(i=0; i<r*fuzziness; i++)
			fuzzymatch[i]=-1;
		fuzzies=0;
		//finds centroids and matches and merges them, returns 1 if still more groups to merge, -1 if not
		//int max_j;
		if (algorithm == 0)
			max_j=compareGroup(blockScore, h_aligned, done, &r, tracer);
		else{
			max_j=compareGroupK(blockScore, h_aligned, done, &r, moreCents, cents, &numCents, tracer);
			if(fuzziness!=0&&max_j!=-1)
				fuzzyK(blockScore, h_aligned, &r, &numCents, fuzzymatch, cents, &fuzziness, &fuzzies, tracer, &tolerance);
    		}
		//printf("MAX J: %d", max_j);
		if(max_j==-1){
			//printf("max_j -1");
			break;
		}
		//printf("after compare group\n");
		//#ifdef CPU
			free(blockScore);
		//#endif
		//continue
		if (algorithm == 1) {
			printf("Number of centroids:%d\n", numCents);
				/*for(i=0; i<numCents; i++)
				printf("Centroid %d: %d Match: %d\n", i, cents[i*2+0], cents[i*2+1]);*/
			printf("Number of fuzzy matches:%d\n", fuzzies);
		}
		//remove groups similar to no other groups
    		//int count;
		w=0;
		//int stop;
		if (algorithm==0) stop = r+1;
		else stop = r;
		for(count=0; count<stop; count++){
			//printf("count: %d\n", count);
			if (algorithm == 1) {
				for(i=0; i<numCents; i++){
					//accounts for centroid matches already removed from list
					if(count==cents[i*2+1]){
						//printf("centmatch: %d\n", cents[i*2+1]);
						w++;
					}
					//int f;
					for(f=0; f<fuzziness; f++){
						if(count==fuzzymatch[i*fuzziness+f]){
							//printf("fuzzymatch: %d\n", fuzzymatch[i*fuzziness+f]);
							w++;
						}
						else if(fuzzymatch[i*fuzziness+f]==-1)
							break;
					}
				}
			}
			else {
				 if (count == max_j) {
					w++;
				}
			}
			//remove group from h_aligned, and add to final_list and update final_groups
			//printf("DONE[%d] = %d\n", count, done[count]);
			if(done[count]==0){
				Node* bad;
				NodeI* badT;
				printf("Removed:(%d)%d\n", count, count-w);
				bad=rmgetnode(h_aligned, count-w);
				badT=rmgetnodeI(tracer, count-w);
				//int groupsize;
				if (count == 0) groupsize = groups[count]+1;
				else groupsize = groups[count]-groups[count-1];
				if(final_list->num_elems > 0)
					final_groups[final_list->num_elems]=final_groups[(final_list->num_elems)-1]+groupsize;
				else
					final_groups[final_list->num_elems]=groupsize-1;
				(final_list->num_elems)++;
				insertend(final_list, bad);
				insertendI(final_tracer, badT);
				free(bad);
				free(badT);
				w++;
				h_aligned->num_elems--;
				tracer->num_elems--;
			}
		}
		//keep r current
		r=h_aligned->num_elems;
		//printf("AFTER BLOCK\n");
		//List_print(h_aligned, SEQ_LENGTH);

		/*printf("Tracer:\n");
		List_printI(tracer);
		printf("Final Tracer:\n");
		List_printI(final_tracer);*/
  	}
     	fclose(data);

	//stop timer
	gettimeofday(&finish, &tz);
     	double clusterElapsed = ( finish.tv_sec - start.tv_sec ) * 1000.0 + ( finish.tv_usec - start.tv_usec ) / 1000.0;
     	//printf("Time to cluster: %lf ms\n", clusterElapsed);
	//printf("Tracer\n");
	//List_printI(tracer);
	//printf("Final tracer\n");
	//List_printI(final_tracer);
/* Design Primers */
 	printf("Designing primers...\n");
}

void GPUDePiCtPlugin::output(std::string file) {
	// flatten final_list and h_aligned
	char* final_aligned = (char*)malloc(ROWS*SEQ_LENGTH*sizeof(char));;
	flatten(final_list->root, final_aligned, NULL, ROWS, SEQ_LENGTH);
      	flatten(h_aligned->root,  aligned, groups, ROWS, SEQ_LENGTH);

	//generate primers for both lists
	if(boolVal==0){
		getprimer(h_aligned, groups, aligned, 0, simacids, codons, SEQ_LENGTH, MIN_PRIMER_LENGTH);
		getprimer(final_list, final_groups, final_aligned, h_aligned->num_elems, simacids, codons, SEQ_LENGTH, MIN_PRIMER_LENGTH);
     	}
	else{
		getNucPrimer(h_aligned, groups, aligned, 0, simNucs, nucCodons, SEQ_LENGTH, MIN_PRIMER_LENGTH);
        	getNucPrimer(final_list, final_groups, final_aligned, h_aligned->num_elems, simNucs, nucCodons, SEQ_LENGTH, MIN_PRIMER_LENGTH);
	}

	//free data
//	free(aligned);
//	free(done);
//	free_list(h_aligned);
//	free_Ilist(tracer);
//	free_Ilist(final_tracer);
//	free(groups);
//	free_list(final_list);
//	free(final_groups);
//	free(cpu_aligned);
//	free(final_aligned);

	#ifndef CPU
		hipFree(gpu_aligned);
		hipFree(gpu_sim);
     		hipFree(gpu_group);
	#endif
}

PluginProxy<GPUDePiCtPlugin> GPUDePiCtPluginProxy = PluginProxy<GPUDePiCtPlugin>("GPUDePiCt", PluginManager::getInstance());


